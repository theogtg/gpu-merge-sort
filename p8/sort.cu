/****************************************************************************************************
* Tyler Griffith                                                                                    *
* December 1st, 2018                                                                                *
* Project 8: Sorting Algorithms on GPU                                                              *
* CSC-4310-01 PROF: R. Shore                                                                        *
* Desc: Use the GPU to merge sort arrays and compare CPU and GPU timings                            *
* To Compile:                                                                                       *
*         nvcc sort.cu -o sort                                                                      *
* To Run:                                                                                           *
*         ./sort <array file>                                                                       *
*****************************************************************************************************/

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <fstream> //for I/O
#include <time.h> //for timing

using namespace std;

//function declarations
int* readArray(char* fileName);
int getSize(char* fileName);
double cuda_merge_sort(int* array, int length, dim3 dimThread, dim3 dimBlock);
void merge(int* array, int left, int mid, int right);
void mergeSort(int* array, int left, int right);
//gpu function declarations
__device__ void cudaMerge(int* a, int* b, int beg, int mid, int end);
__global__ void cudaMergeSort(int* a, int* b, int width, int length, int pieces, dim3* thread, dim3* block);
__device__ unsigned int getThreadIdx(dim3* thread, dim3* block);

//global find min function
#define min(a, b) (a < b ? a : b)

//driver
int main(int argc, char* argv[]){
   //variable declaration
   int size;
   dim3 dimThread, dimBlock; //threads per block and blocks per grid;

   //set initial values for testing
   dimThread.x = 32;
   dimThread.y = 1;
   dimThread.z = 1;
   dimBlock.x = 8;
   dimBlock.y = 1;
   dimBlock.z = 1;

   //make sure correct syntax is used
   if(argc != 2){
      cout << "Error! You do not have an element to your command!" << endl;
      cout << "To sort an array please use the following syntax:" << endl;
      cout << "./sort <array file>" << endl;
      return -1;
   }

   //read in size of the array
   size = getSize(argv[1]);
   int* gpuArray = new int[size];
   //read in array
   gpuArray = readArray(argv[1]);

	//created different array so both could be sorted
	int* cpuArray = new int[size];
	cpuArray = gpuArray;

	//GPU timing
	double gpuDuration = cuda_merge_sort(gpuArray, size, dimThread, dimBlock);

	//output sorted GPU array to separate file
	ofstream gpuSortedArray("gpuSortedArray");
	for(int i=0; i<size; i++){
		gpuSortedArray << gpuArray[i] << " ";
	}
	gpuSortedArray.close();

	//CPU timing
	clock_t cpuStart = clock();

	//call cpu merge sort function
	mergeSort(cpuArray, 0, size-1);

	//CPU timing
	clock_t cpuEnd = clock();
	double cpuDuration = (double)(cpuEnd-cpuStart)/CLOCKS_PER_SEC;

	//print timing
	cout << "gpu sort completed in " << gpuDuration*1000 << " milliseconds!" << endl;
	cout << "cpu sort completed in " << cpuDuration*1000 << " milliseconds!" << endl;

   return 0;
}

/****************************************************
 * Function: getSize - fetch and return the size    *
 *                     of the array from I/O        *
 * precondition: filename is fetched from argv[1]   *
 *               in the main                        *
 * postcondition: the size of the array is          *
 *                returned to main                  *
 ****************************************************/
int getSize(char* fileName){
	//variable delcaration
   int size;
	//initialize and open file
   ifstream inFile(fileName);
   if(inFile.is_open()){
	   //read in size
      inFile >> size;
   }
  //close file
  inFile.close();
  return size;
}

/****************************************************
 * Function: readArray - fetch and return the array *
 *                       from I/O                   *
 * precondition: filename is fetched from argv[1]   *
 *               in the main                        *
 * postcondition: the array is returned to main     *
 ****************************************************/
int* readArray(char* fileName){
   //variable declaration
   int size;
   ifstream inFile(fileName);

   //read in array size
   inFile >> size;

   //allocate memory
   int *array = new int[size];
   
   //read in array
   for(int i=0; i<size; i++)
      inFile >> array[i];

   //close file and return array
   inFile.close();
   return array;
}

/****************************************************
 * Function: mergeSort - merge sort the given array *
 *                       on the CPU                 *
 * precondition: - array is established in the main *
 *                 through readArray()              *
 *               - right is established through     *
 *                 getsize()                        *
 * postcondition: the array is sorted               *
 ****************************************************/
void mergeSort(int* array, int left, int right){
	//left is left index right is right index
	if(left < right){
		//same as (left+right)/2, but avoids overflow for large nums
		int mid = left+(right-left)/2;

		//sort first and seconds halves
		mergeSort(array, left, mid);
		mergeSort(array, mid+1, right);
		//merge the pieces together
		merge(array, left, mid, right);
	}
}

/****************************************************
 * Function: merge - merges two subarrays           *
 * precondition: - array is established in the main *
 *                 through readArray()              *
 *               - right is established through     *
 *                 getsize()                        *
 *               - mid is established in mergeSort()*
 * postcondition: the two subarrays are merged      *
 ****************************************************/
void merge(int* array, int left, int mid, int right){
	//variable declaration
	int i,j,k;
	int n1 = mid-left+1;
	int n2 = right-mid;

	//temp arrays
	int L[n1], R[n2];

	//copy data to temp arrays
	for(i=0; i<n1; i++)
		L[i] = array[left+i];
	for(j=0; j<n2; j++)
		R[j] = array[mid+1+j];

	//merge temp arrays back
	i=0;//index for first array
	j=0;//index for second array
	k=left;//index for merged array

	while(i<n1 && j<n2){
		if(L[i]<=R[j]){
			array[k] = L[i];
			i++;
		}
		else{
			array[k] = R[j];
			j++;
		}
		k++;
	}

	//copy remaining elements of L
	while(i<n1){
		array[k] = L[i];
		i++;
		k++;
	}

	//copy remaining elements of R
	while(j<n2){
		array[k] = R[j];
		j++;
		k++;
	}
}

/***********************************************************
 * Function: cuda_merge_sort - allocates memory on GPU for *
 *                             array then copies the array *
 *                             to the GPU and calls the    *
 *                             GPU merge sort function     *
 * precondition: - array is established in the main        *
 *                 through readArray()                     *
 *               - length is established in the main       *
 *                 through getSize()                       *
 *               - dimThread and dimBlock are established  *
 *                 in the main                             *
 * postcondition: the array is sorted on the GPU and the   *
 *                time taken to sort the array on the      *
 *                GPU is returned back to main             *
 ***********************************************************/
double cuda_merge_sort(int* array, int length, dim3 dimThread, dim3 dimBlock){
	//variable declaration
   int* dArray;
   int* dSwap;
   dim3* dThread;
   dim3* dBlock;

   //allocate memory for array on gpu
   hipMalloc((void**)&dArray, length*sizeof(int));
   hipMalloc((void**)&dSwap, length*sizeof(int));

   //copy array to gpu
   hipMemcpy(dArray, array, length*sizeof(int), hipMemcpyHostToDevice);

   //allocate memory for the thread and block info on gpu
   hipMalloc((void**)&dThread, sizeof(dim3));
   hipMalloc((void**)&dBlock, sizeof(dim3));

   //copy thread and block info to gpu
   hipMemcpy(dThread, &dimThread, sizeof(dim3), hipMemcpyHostToDevice);
   hipMemcpy(dBlock, &dimBlock, sizeof(dim3), hipMemcpyHostToDevice);

   //for copying
   int* x = dArray;
   int* y = dSwap;
   
   //get thread count
   int threadCount = dimThread.x * dimThread.y * dimThread.z * dimBlock.x * dimBlock.y * dimBlock.z;

   //timing
	clock_t start = clock();

   //cut the array into different pieces and give those pieces to each thread
   for(int width = 2; width < (length << 1); width <<= 1){
	   //variable delcaration
      int pieces = length / ((threadCount) * width) + 1;

      //call the sort
      cudaMergeSort<<<dimBlock, dimThread>>>(x, y, width, length, pieces, dThread, dBlock);

      //can swap the input/output arrays instead of copying
      x = x == dArray ? dSwap : dArray;
      y = y == dArray ? dSwap : dArray;
      
   }

	//timing
	clock_t end = clock();
	double duration = (double)(end-start)/CLOCKS_PER_SEC;

   //retrieve array from the GPU
   hipMemcpy(array, dArray, length*sizeof(int), hipMemcpyDeviceToHost);

   //free memory
   hipFree(dArray);
   hipFree(dSwap);

	return duration;
}

/*************************************************************
 * Global Function: cudaMergeSort - merge sorts the given    *
 *                                  array on the GPU         *
 * precondition: - a is established in cuda_merge_sort()     *
 *               - b is established in cuda_merge_sort()     *
 *               - width is established in cuda_merge_sort() *
 *               - length is established in getSize()        *
 *               - pieces is established in cuda_merge_sort()*
 *               - thread and block are both established     *
 *                 within the main                           *
 * postcondition: the array is sorted on the GPU             *
 *************************************************************/
__global__ void cudaMergeSort(int* a, int* b, int width, int length, int pieces, dim3* thread, dim3* block){
   //set the thread index
   unsigned int idx = getThreadIdx(thread, block);
   //initialize positioning
   int beg = width * idx * pieces;
   int mid, end;

	//for each of the pieces
   for(int piece=0; piece<pieces; piece++){
      if (beg >= length)
         break;
		//initialize the middle of the array
      mid = min(beg+(width>>1), length);
	   //initialize the end of the array
      end = min(beg+width, length);
	   //merge
      cudaMerge(a, b, beg, mid, end);
	   //update the beginning of the array
      beg += width;
   }
}

/*************************************************************
 * Device Function: cudaMerge - merges two subarrays         *
 * precondition: - a is established in cuda_merge_sort()     *
 *               - b is established in cuda_merge_sort()     *
 *               - beg, mid and end are established within   *
 *                 the cudaMergeSort global function         *
 * postcondition: the two subarrays are merged together      *
 *************************************************************/
__device__ void cudaMerge(int* a, int* b, int beg, int mid, int end){
	//initialize positioning
   int i = beg;
   int j = mid;

	//loop through the chunk
   for(int k=beg; k<end; k++){
	   //swap
      if(i < mid && (j>=end || a[i]<a[j])){
         b[k] = a[i];
         i++;
      } else {
         b[k] = a[j];
         j++;
      }
   }
}

/*************************************************************
 * Device Function: getThreadIdx - fetches the current       *
 *                                 thread index              *
 * precondition: - thread and block are established within   *
 *                 the main                                  *
 * postcondition: the current thread index is returned       *
 *************************************************************/
__device__ unsigned int getThreadIdx(dim3* thread, dim3* block) {
   int x;
   //calculates and returns the index of the current thread
   return threadIdx.x +
          threadIdx.y * (x  = thread->x) +
          threadIdx.z * (x *= thread->y) +
          blockIdx.x  * (x *= thread->z) +
          blockIdx.y  * (x *= block->z) +
          blockIdx.z  * (x *= block->y);
}